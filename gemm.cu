#include <iostream>
#include <cstdlib>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>

double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return((double)tp.tv_sec+(double)tp.tv_usec*1e-6);
}

#define EPS 1e-4
typedef struct matrix
{
    size_t height;
    size_t width;    /* data */
    float* data; 
} Matrix;

/*
 * CPU 矩阵相关函数
 */ 

void CHECK_STATUS(bool checkState, const char* info) {
    if (!checkState) {
        printf("ERROR:%s", info);
        exit(-1);
    }
}

void SetMatValCPU(Matrix *m, int rowIdx, int colIdx, float val) {
    m->data[rowIdx * (m -> width) + colIdx] = val;
}

float GetMatValCPU(Matrix *m, int rowIdx, int colIdx) {
    return m->data[rowIdx * (m -> width) + colIdx];
}

void GemmCPU(Matrix* m1, Matrix *m2, Matrix *out) {
    CHECK_STATUS(m1->width == m2->height, "gemm invlid size");
    double start = cpuSecond();
    for (size_t i = 0; i < m1 -> height; i++) {
        for (size_t k = 0; k < m2 -> width; k++) {
            float sumVal = 0.0;
            for (size_t j = 0; j < m2 -> width; j++) {
                sumVal += GetMatValCPU(m1, i, j) * GetMatValCPU(m2, j, k);
            }
            SetMatValCPU(out, i, k, sumVal);
        }
    }
    double end = cpuSecond();
    std::cout << "GemmCPU elapse:" << end - start << std::endl;
}

Matrix* NewMatrixCPU(size_t height, size_t width) {
    Matrix* mat = (Matrix*)malloc(sizeof(Matrix));
    CHECK_STATUS(mat != nullptr, "malloc ERR");
    mat -> height = height;
    mat -> width = width;
    mat -> data = (float*)malloc(sizeof(float) * width * height);
    CHECK_STATUS(mat -> data != nullptr, "malloc data ERR");
    return mat;
}

void FreeMatrixCPU(Matrix* m1) {
    free(m1 -> data);
    free(m1);
}


void RandomFillMatrixCPU(Matrix* m1, float B = 100.0, float A = 0.0) {
    srand (static_cast <unsigned> (time(0)));
    for (size_t i = 0; i < m1 -> height; i++) {
        for (size_t j = 0; j < m1 -> width; j++) {
            float randomVal = A + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(B-A)));
            SetMatValCPU(m1, i, j, randomVal);
        }
    }
}


void NormalFillMatrixCPU(Matrix* m1) {
    for (size_t i = 0; i < m1 -> height; i++) {
        for (size_t j = 0; j < m1 -> width; j++) {
            SetMatValCPU(m1, i, j, i * m1 -> width * m1 -> height + j);
        }
    }
}

void PrintMatrix(Matrix* m1, std::string info) {
    std::cout << "==========" << info << "==============\n";
    std::cout << "np.array([\n";
    for (size_t i = 0; i < m1 -> height; i++) {
        std::cout << "[";
        for (size_t j = 0; j < m1 -> width; j++) {
            std::cout << GetMatValCPU(m1, i, j) << ",";
        }
        std::cout << "],\n";
    }
    std::cout << "])\n";
    std::cout << std::endl;
}

bool CompareMatrixCPU(Matrix* m1, Matrix* m2) {
    if (m1 -> height != m2 -> height || m1 -> width != m2 -> width) {
        return false;
    }
    float diff = 0.0;
    for (size_t i = 0; i < m1 -> height; i++) {
        for (size_t j = 0; j < m1 -> width; j++) {
            //std::cout << GetMatValCPU(m1, i, j) << " || "<< GetMatValCPU(m2, i, j) << std::endl;
            diff += abs(GetMatValCPU(m1, i, j) - GetMatValCPU(m2, i, j));
        }
    }
    std::cout << "diff value:" << diff << std::endl;
    return diff < EPS;
}


/*
 * GPU 矩阵相关函数
 */

Matrix* NewMatrixGPUFromCPU(Matrix* cpuMat) {
    Matrix* mat;
    float* d_data;
    CHECK_STATUS(hipSuccess == hipMalloc((void**)&mat, sizeof(Matrix)), "cuda mat malloc failed");
    size_t data_size = sizeof(float) * cpuMat->height * cpuMat->width;
    //std::cout << "data_size:" << data_size << std::endl;

    CHECK_STATUS(hipSuccess == hipMalloc((void**)&d_data, data_size), "cuda data malloc failed");
    CHECK_STATUS(hipSuccess == hipMemcpy(d_data, cpuMat->data, data_size, hipMemcpyHostToDevice), "cuda copy failed");

    float* tmp_h_data = cpuMat -> data;
    cpuMat -> data = d_data;
    CHECK_STATUS(hipSuccess == hipMemcpy(mat, cpuMat, sizeof(Matrix), hipMemcpyHostToDevice), "cuda copy failed");
    cpuMat -> data = tmp_h_data;

    return mat;
}

Matrix* NewMatrixGPU(size_t height, size_t width) {
    Matrix tmpHostMat;
    Matrix* tmpHostMatPtr = &tmpHostMat;

    tmpHostMatPtr -> height = height;
    tmpHostMatPtr -> width = width;
    tmpHostMatPtr -> data = nullptr;
    Matrix* mat;
    CHECK_STATUS(hipSuccess == hipMalloc((Matrix**)&mat, sizeof(Matrix)), "gpu malloc failed");
    size_t data_size = sizeof(float) * width * height;
    CHECK_STATUS(hipSuccess == hipMalloc((float**)&(tmpHostMatPtr -> data), data_size),  "gpu malloc data failed");
    CHECK_STATUS(hipSuccess == hipMemcpy(mat, tmpHostMatPtr, sizeof(Matrix), hipMemcpyHostToDevice), "cuda copy failed");
    return mat;
}

Matrix* NewMatrixCPUFromGPU(Matrix* mGpu) {
    Matrix* mCpu = (Matrix*)malloc(sizeof(Matrix));// ? 这里可以直接这么拷贝吗?
    CHECK_STATUS(hipSuccess == hipMemcpy(mCpu, mGpu, sizeof(Matrix), hipMemcpyDeviceToHost), "cuda gpu to cpu failed");
    std::cout << "cpu copy done: width=" << mCpu -> width << " height="<< mCpu -> height << std::endl;
    size_t dataSize = sizeof(float) * mCpu -> width * mCpu -> height;
    float* cpuData = (float*)malloc(dataSize);
    std::cout << "cpuData done" << std::endl;
    CHECK_STATUS(hipSuccess == hipMemcpy(cpuData, mCpu -> data, dataSize, hipMemcpyDeviceToHost), "cuda cpu to gpu failed");
    mCpu -> data = cpuData;
    std::cout << "hipMemcpy done" << std::endl;
    return mCpu;
}

void FreeMatrixGPU(Matrix* m1) {
    Matrix tmpFree;
    CHECK_STATUS(hipSuccess == hipMemcpy(&tmpFree, m1, sizeof(Matrix), hipMemcpyDeviceToHost), "cuda gpu to cpu failed");
    hipFree(tmpFree.data);
    hipFree(m1);
}

__device__ void SetMatValGPU(Matrix *m, int rowIdx, int colIdx, float val) {
    m->data[rowIdx * (m -> width) + colIdx] = val;
    //m->data[0] = val;
}

__device__ float GetMatValGPU(Matrix *m, int rowIdx, int colIdx) {
    return m->data[rowIdx * (m -> width) + colIdx];
}

__global__ void GemmGPUFunc(Matrix* A, Matrix* B, Matrix* C) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    //printf("blockIdx.x=%d blockIdx.y=%d threadIdx.x=%d threadIdx.y=%d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
    //printf("matIdx=(%02d, %02d)\n", i, k);
    //C->data[i * (C -> width) + k] = 1.0;
    //setMatVal(C, i, k, 5.0);
    if (i < A->height && k < B->width) {
        float res = 0;
        for (int j = 0 ; j < (A->width); j++) {
            //printf("A[%d][%d]=%f B[%d][%d]=%f\n", i, j, getMatVal(A, i, j), j, k, getMatVal(B, j, k));
            res += GetMatValGPU(A, i, j) * GetMatValGPU(B, j, k);
        }
        SetMatValGPU(C, i, k, res);
    }
}

void GemmGPU(Matrix* m1, Matrix *m2, Matrix *out) {
    //CHECK_STATUS(m1->width == m2->height, "gemm invlid size");
    int BLOCK_X = 32; int BLOCK_Y = 32;
    //int GRID_X = (m2 -> width + BLOCK_X - 1) / BLOCK_X; int GRID_Y = (m1 -> height + BLOCK_Y - 1) / BLOCK_Y;
    //printf("GRID_X=%d, GRID_Y=%d", GRID_X, GRID_Y);

    dim3 blocks(BLOCK_X, BLOCK_Y);
    dim3 grids(5, 5);// kernel调用

    double start = cpuSecond();
    GemmGPUFunc<<<grids, blocks>>>(m1, m2, out);
    hipDeviceSynchronize();
    double end = cpuSecond();
    std::cout << "GemmGPU elapse:" << end - start << std::endl;
}

void GPUTest() {
    bool printDebugFlag = true;
    // mat1 = np.array([[1, 2, 3], [1, 2, 3], [1, 2, 3]])

    Matrix* m1CPU = NewMatrixCPU(3, 3);
    Matrix* m2CPU = NewMatrixCPU(3, 3);
    Matrix* mResCPU = NewMatrixCPU(3, 3); // 标准答案
    //Matrix* mResCPU = NewMatrixCPU(3, 3); // 计算答案

    RandomFillMatrixCPU(m1CPU);
    RandomFillMatrixCPU(m2CPU);
    //RandomFillMatrixCPU();
    GemmCPU(m1CPU, m2CPU, mResCPU);

    Matrix* m1GPU = NewMatrixGPUFromCPU(m1CPU);
    //std::cout << "malloc gpu1" << std::endl;

    Matrix* m2GPU = NewMatrixGPUFromCPU(m2CPU);
    //std::cout << "malloc gpu2" << std::endl;

    Matrix* mResGPU = NewMatrixGPU(m1CPU->height, m2CPU->width); // 计算答案
    //std::cout << "malloc gpu3" << std::endl;

    GemmGPU(m1GPU, m2GPU, mResGPU);
    //std::cout << "GemmGPU done" << std::endl;

    Matrix* mResCPULoad = NewMatrixCPUFromGPU(mResGPU);

    if (printDebugFlag) {
        PrintMatrix(m1CPU, "m1CPU");
        PrintMatrix(m2CPU, "m2CPU");
        PrintMatrix(mResCPU, "mResCPU");
        PrintMatrix(mResCPULoad, "mResCPULoad");
    }
    
    bool cmpRes = CompareMatrixCPU(mResCPU, mResCPULoad);
    FreeMatrixCPU(m1CPU);
    FreeMatrixCPU(m2CPU);
    //FreeMatrixCPU(stdmRes);
    FreeMatrixCPU(mResCPU);
    FreeMatrixCPU(mResCPULoad);
    
    //std::cout << "cpu freed" << std::endl;
    FreeMatrixGPU(m1GPU);
    FreeMatrixGPU(m2GPU);
    FreeMatrixGPU(mResGPU);
    //std::cout << "gpu freed" << std::endl;

    if (cmpRes) {
        std::cout << "INFO: gemm Success" <<  std::endl;
    } else {
        std::cout << "ERROR: gemm Failed" << std::endl;
    }
    
}

int main() {
    GPUTest();
}