#include <iostream>

template <typename T>
typedef struct matrix
{
    int width;    /* data */
    int height;
    T* data; 
} Matrix;

/*
 *矩阵相关函数
 * 
 */
void InitMatrix(Matrix **matp, int height, int width) {
    hipMallocManaged((void**)matp, sizeof(Matrix));
    Matrix *mat = *matp;
    mat -> width = width;  mat -> height = height;
    hipMallocManaged((void**)&(mat->data), (mat->width) * (mat->height) * sizeof(float));
    return;
}

void DisplayMatrix(Matrix* mat) {
    std::cout << "=======" << std::endl;
    for (int i = 0; i < mat -> height; i++) {
        for (int j = 0; j < mat -> width; j++) {
            std::cout << mat -> data[i * mat -> width + j] << ",";
        }
        std::cout << std::endl;
    }
}

void FreeMatrix(Matrix* mat) {
    hipFree(mat -> data); // 释放内存
    hipFree(mat); // 释放内存
}

void GenerateRandomMatrix() {

}

void MatrixMultiplyTest() {
    const int I = 3;
    const int J = 4;
    const int K = 5;
    const int BLOCK_SIZE = 256;
    Matrix *matA, *matB, *matC;

    InitMatrix(&matA, I, J); FillMatrix(matA, 1.0);
    InitMatrix(&matB, J, K); FillMatrix(matB, 2.0);
    InitMatrix(&matC, I, K); FillMatrix(matC, 0.0);

    DisplayMatrix(matA);
    DisplayMatrix(matB);
    DisplayMatrix(matC);

    //std::cout << "init fin";
    int BLOCK_X = 32; int BLOCK_Y = 32;
    int GRID_X = (K + BLOCK_X - 1) / BLOCK_X; int GRID_Y = (I + BLOCK_Y - 1) / BLOCK_Y;
    printf("GRID_X=%d, GRID_Y=%d", GRID_X, GRID_Y);

    dim3 blocks(BLOCK_X, BLOCK_Y);
    dim3 grids(BATCH_SIZE, NUM_HEAD);// kernel调用
    MatMultiply<<<grids, blocks>>>(matA, matB, matC);
    
    // 必须有这句话
    hipDeviceSynchronize();
    DisplayMatrix(matC);
    // 在原来的数据里面进行读取就可以了。
    FreeMatrix(matA); // 释放内存
    FreeMatrix(matB); // 释放内存
    FreeMatrix(matC); // 释放内存
}

int main() {
    MatrixMultiplyTest();
    return 0;
}